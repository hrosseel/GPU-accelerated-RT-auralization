#include "hip/hip_runtime.h"
using namespace torch::indexing;
#include <torch/extension.h>
#include <c10/cuda/CUDAException.h>

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

#ifndef NUM_CHANNELS
    #error "NUM_CHANNELS is not defined"
#endif
#ifndef BLOCK_SIZE
    #error "BLOCK_SIZE is not defined"
#endif
#ifndef NUM_PARTS
    #error "NUM_PARTS is not defined"
#endif

#define NUM_THREADS 256

inline unsigned int cdiv(unsigned int a, unsigned int b) { return (a + b - 1) / b;}

__global__ void conv_kernel(const c10::complex<float>* fdl, const c10::complex<float>* filters_fd, int fdl_cursor, c10::complex<float>* output_fd) {
    
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (thread_id >= NUM_CHANNELS * (BLOCK_SIZE + 1)) return;

    const int channel_id = thread_id / (BLOCK_SIZE + 1);
    const int bin_id = thread_id % (BLOCK_SIZE + 1);
    int cursor = fdl_cursor;

    const int fdl_offset = bin_id * NUM_PARTS;
    const int filter_offset = channel_id * ((BLOCK_SIZE + 1) * NUM_PARTS) + bin_id * NUM_PARTS;
    const int output_offset = channel_id * (BLOCK_SIZE + 1) + bin_id;

    c10::complex<float> out = 0;
    for (int k = 0; k < NUM_PARTS; ++k) {
        out += fdl[fdl_offset + cursor] * filters_fd[filter_offset + k];
        cursor = (cursor - 1 + NUM_PARTS) % NUM_PARTS;
    }
    output_fd[output_offset] = out;
}

__global__ void conv_kernel_multi(const c10::complex<float>* fdl, const c10::complex<float>* filters_fd, int fdl_cursor, c10::complex<float>* output_fd) {
    
    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (thread_id >= NUM_CHANNELS * (BLOCK_SIZE + 1)) return;

    const int channel_id = thread_id / (BLOCK_SIZE + 1);
    const int bin_id = thread_id % (BLOCK_SIZE + 1);
    int cursor = fdl_cursor;

    const int filter_offset = channel_id * ((BLOCK_SIZE + 1) * NUM_PARTS) + bin_id * NUM_PARTS;
    const int output_offset = channel_id * (BLOCK_SIZE + 1) + bin_id;

    c10::complex<float> out = 0;
    for (int k = 0; k < NUM_PARTS; ++k) {
        out += fdl[filter_offset + cursor] * filters_fd[filter_offset + k];
        cursor = (cursor - 1 + NUM_PARTS) % NUM_PARTS;
    }
    output_fd[output_offset] = out;
}

torch::Tensor part_conv_gpu(torch::Tensor input_fd, torch::Tensor fdl, torch::Tensor filters_fd, int fdl_cursor) {
    CHECK_INPUT(input_fd);
    CHECK_INPUT(fdl);
    CHECK_INPUT(filters_fd);

    auto output_fd = torch::empty({NUM_CHANNELS, BLOCK_SIZE+1}, input_fd.options());

    int threads = 256;
    int blocks = cdiv(NUM_CHANNELS * (BLOCK_SIZE + 1), threads);

    // Store the fd signal in a frequency-domain delay line
    fdl.index_put_({Slice(), Slice(0, BLOCK_SIZE + 1), fdl_cursor}, input_fd);

    if (fdl.dim() == 3 && fdl.sizes()[0] == 1) {
        conv_kernel<<<blocks, threads>>>(fdl.data_ptr<c10::complex<float>>(), filters_fd.data_ptr<c10::complex<float>>(), fdl_cursor, output_fd.data_ptr<c10::complex<float>>());

    } else if (fdl.dim() == 3 && fdl.sizes()[0] > 1) {
        conv_kernel_multi<<<blocks, threads>>>(fdl.data_ptr<c10::complex<float>>(), filters_fd.data_ptr<c10::complex<float>>(), fdl_cursor, output_fd.data_ptr<c10::complex<float>>());

    } else {
        throw std::runtime_error("Invalid fdl size");
    }

    C10_CUDA_KERNEL_LAUNCH_CHECK(); // Check for errors
    return output_fd;
}